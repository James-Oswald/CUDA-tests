#include "hip/hip_runtime.h"

#include<iostream>
#include<thrust/device_vector.h>

__global__ void sumRow(double* vecStart, double* vec2Start){
    vecStart[blockIdx.x] = vecStart[blockIdx.x] + vec2Start[blockIdx.x];
}

double sum(const std::vector<double>& vec){
    thrust::device_vector<double> dv(vec.begin(), vec.end());
    while(dv.size() > 1){
        int newSize = dv.size()/2+(dv.size() % 2 == 1);
        double* dvloc = dv.data().get();
        sumRow<<<dv.size()/2, 1>>>(dvloc, dvloc+newSize);
        hipDeviceSynchronize();
        dv.resize(newSize);
    }
    return dv[0];
}

int main(){
    std::vector<double> data;
    for(int i = 0; i < 100; i++)
        data.push_back(0.5);
    std::cout<<sum(data)<<std::endl;
}