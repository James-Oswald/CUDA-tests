#include "hip/hip_runtime.h"

#include<iostream>
#include<thrust/device_vector.h>

__global__ void sumRow(double* vecStart, double* vec2Start){
    vecStart[blockIdx.x] = vecStart[blockIdx.x] + vec2Start[blockIdx.x];
}

double sum(const std::vector<double>& vec){
    thrust::device_vector<double> dv(vec.begin(), vec.end());
    while(dv.size() > 1){
        bool odd = dv.size() % 2 == 1;
        double* dvloc = dv.data().get();
        sumRow<<<dv.size()/2, 1>>>(dvloc, dvloc+(dv.size()/2+(odd?1:0)));
        hipDeviceSynchronize();
        dv.resize(dv.size()/2+(odd?1:0));
    }
    return dv[0];
}

int main(){
    std::vector<double> data = {.5, .5, .5, .5, .9};
    std::cout<<sum(data)<<std::endl;
}