#include "hip/hip_runtime.h"

#include<cstdio>
#include<cassert>
#include<iostream>
#include<thrust/device_vector.h>

__global__ void sum(double* data, int size){
    assert(size <= gridDim.x);
    if(size == 1) 
        return;
    int half = size / 2;
    int newSize = half + (size % 2 == 1);    
    if(blockIdx.x >= half) 
        return;
    data[blockIdx.x] = data[blockIdx.x] + data[blockIdx.x + newSize];
    if(blockIdx.x == 0){
        for(int i = 0; i < size; i++)
            printf("%02f ", data[i]);
        printf("\n%d | %d\n", size, newSize);
        sum<<<newSize,1>>>(data, newSize);
    }
}

int main(){
    std::vector<double> data;
    for(int i = 0; i < 100; i++)
        data.push_back(0.5);
    thrust::device_vector<double> dv(data.begin(), data.end());
    sum<<<dv.size(),1>>>(dv.data().get(), dv.size());
    std::cout<<dv[0]<<std::endl;
}

