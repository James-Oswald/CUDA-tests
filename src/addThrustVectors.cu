#include "hip/hip_runtime.h"

#include<iostream>
#include <thrust/copy.h>
#include<thrust/device_vector.h>

__global__ void add(const double* in1, const double* in2, double* output){
    output[threadIdx.x] = in1[threadIdx.x] + in2[threadIdx.x];
}

int main(){
    std::vector<double> data = {10, 20, 30, 40};
    thrust::device_vector<double> in1(data.begin(), data.end());
    thrust::device_vector<double> in2(data.begin(), data.end());
    thrust::device_vector<double> out(data.size());
    add<<<1, data.size()>>>(in1.data().get(), in2.data().get(), out.data().get());
    std::vector<double> res(out.size());
    thrust::copy(out.begin(), out.end(), res.begin());
    for(int i = 0; i < 4; i++)
        std::cout<<res[i]<<std::endl;
}