
#include <hip/hip_runtime.h>
#include<cstdio>
#include<iostream>
#include<vector>

#define vectorSize 10

__global__ void add(const double* in1, const double* in2, double* output){
    output[threadIdx.x] = in1[threadIdx.x] + in2[threadIdx.x];
}

int main(){
    std::vector<double> ins = {1, 1, 1, 1, 1};
    int inSize = ins.size()*sizeof(double);
    std::vector<double> result;
    result.resize(ins.size());
    double *din1, *din2, *dout;
    hipMalloc((void**)&din1, inSize);
    hipMalloc((void**)&din2, inSize);
    hipMalloc((void**)&dout, inSize);
    hipMemcpy((void*)din1, (void*)ins.data(), inSize, hipMemcpyHostToDevice);
    hipMemcpy((void*)din2, (void*)ins.data(), inSize, hipMemcpyHostToDevice);
    add<<<1, ins.size()>>>(din1, din2, dout);
    hipDeviceSynchronize();
    hipMemcpy((void*)result.data(), dout, inSize, hipMemcpyDeviceToHost);
    hipFree(din1); hipFree(din2); hipFree(dout);
    for(double elm : result)
        std::cout<<elm<<std::endl;
    return 0;
}