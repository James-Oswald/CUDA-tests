

#include <hip/hip_runtime.h>
#include<cstdio>
#include<iostream>
#include<vector>

#define vectorSize 10

__global__ void sum(const double* in1, const double* in2, double* output){
    printf("Thread: %d\n", threadIdx.x);
    output[threadIdx.x] = in1[threadIdx.x] + in2[threadIdx.x];
}

int main(){
    int rt, dv;
    hipError_t c2 = hipDriverGetVersion(&dv);
    hipError_t c1 = hipRuntimeGetVersion(&rt);
    std::cout<<"CUDA RT:"<<c1<<":"<<rt<<"\nCUDA Dv:"<<c2<<":"<<dv<<std::endl;
    int inSize = vectorSize*sizeof(double);
    std::vector<double> ins, result;
    result.resize(vectorSize);
    for(int i = 0; i < vectorSize; i++)
        ins.push_back(i);
    double* din1, *din2, *dout;
    hipMalloc((void**)&din1, inSize);
    hipMalloc((void**)&din2, inSize);
    hipMalloc((void**)&dout, inSize);
    hipMemcpy((void*)din1, (void*)ins.data(), inSize, hipMemcpyHostToDevice);
    hipMemcpy((void*)din2, (void*)ins.data(), inSize, hipMemcpyHostToDevice);
    sum<<<1, 30>>>(din1, din2, dout);
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
    hipMemcpy((void*)result.data(), dout, inSize, hipMemcpyDeviceToHost);
    hipFree(din1); hipFree(din2); hipFree(dout);
    for(double elm : result)
        std::cout<<elm<<std::endl;
    return 0;
}